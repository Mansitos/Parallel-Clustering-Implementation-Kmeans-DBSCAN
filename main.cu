﻿/*
Progetto Programmazione su Architetture Parallele - UNIUD 2021
Mansi Andrea & Christian Cagnoni
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "kmeans.h"
#include "inputgenerator.h"
#include <random>
#include <iostream>
#include "utils.h"
#include "tester.h"

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char* file, int line);

int main() {
	runTestSession(true);
}

/*
Handles CUDA Errors and print them.
*/
static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

