#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <iostream>
#include "utils.h"

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char* file, int line);

__device__ float euclideanDistance(float* dataPoint, float* centroid, int dim);

__device__ int calculateCentroid(float* dataPoint, int dim, int k, float** centroids);

__device__ bool convergenceCheck = true;
__device__ unsigned int countB = 0;

__device__ double atomicAddDouble(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

static __inline__ __device__ bool atomicCASBool(bool* address, bool compare, bool val)
{
	unsigned long long addr = (unsigned long long)address;
	unsigned pos = addr & 3;  // byte position within the int
	int* int_addr = (int*)(addr - pos);  // int-aligned address
	int old = *int_addr, assumed, ival;

	bool current_value;

	do
	{
		current_value = (bool)(old & ((0xFFU) << (8 * pos)));

		if (current_value != compare) // If we expected that bool to be different, then
			break; // stop trying to update it and just return it's current value

		assumed = old;
		if (val)
			ival = old | (1 << (8 * pos));
		else
			ival = old & (~((0xFFU) << (8 * pos)));
		old = atomicCAS(int_addr, assumed, ival);
	} while (assumed != old);

	return current_value;
}

__device__ bool AtomicBool(bool* address, bool val)
{
	// Create an initial guess for the value stored at *address.
	bool guess = *address;
	bool oldValue = atomicCASBool(address, guess, val);

	// Loop while the guess is incorrect.
	while (oldValue != guess)
	{
		guess = oldValue;
		oldValue = atomicCASBool(address, guess, val);
	}
	return oldValue;
}

__device__ double euclideanDistance_device(double* dataPoint, int tid, double* centroid, int index, int dim) {
	double sum = 0;
	for (int i = 0; i < dim; i++) {
		sum += pow((dataPoint[tid + i] - centroid[index + i]), 2);
	}
	double distance = sqrt(sum);
	return distance;
}

__device__ int calculateCentroid_device(double* dataPoint, int tid, int dim, int k, double* centroids) {
	int nearestCentroidIndex = dataPoint[tid + dim];
	bool firstIteration = true;
	double minDistance = 0;
	for (int i = 0; i < k; i++) {
		double distance = euclideanDistance_device(dataPoint, tid, centroids, i * dim, dim);
		if (firstIteration) {
			firstIteration = false;
			nearestCentroidIndex = i;
			minDistance = distance;
		}
		else if (distance < minDistance ){
			nearestCentroidIndex = i;
			minDistance = distance;
		}
	}
	return nearestCentroidIndex;
}

__global__ void k_means_cuda_device_update_centroids(double* d_dataPoints, double* d_centroids, int* assignedPoints, int length, int dim, int k, int NumBlocks) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= length * (dim + 1))
		return;
	if (threadIdx.x < k && countB == 0)
		assignedPoints[tid] = 0;
	int tmp = tid * k;
	int tmpId = threadIdx.x * k;
	if (tmpId < k * dim && countB == 0) {
		for (int i = tmpId; i < tmpId + dim; i++)
			d_centroids[i] = 0;
	}

	if (threadIdx.x == 0)
		atomicAdd(&countB, 1);

	__syncthreads();

	extern __shared__ double s[];
	double* b_centroids = s;
	int* b_assignedPoints = (int*)&b_centroids[k * (dim + 1)];
	if (threadIdx.x == 0)
		for (int i = 0; i < k; i++) {
			b_assignedPoints[i] = 0;
			for (int j = 0; j < dim; j++)
				b_centroids[(i * dim) + j] = 0;
		}
	__syncthreads();

	if (tid % (dim + 1) == 0) {//prima cordinata
		atomicAddDouble(&b_centroids[((int)d_dataPoints[tid + dim] * k)], d_dataPoints[tid]);
	}
	else if ((tid + 1) % (dim + 1) != 0) {//seconda
		atomicAddDouble(&b_centroids[((int)d_dataPoints[tid + (tid % (dim + 1))] * k) + (tid % (dim + 1))], d_dataPoints[tid]);
	}
	else {//cluster
		atomicAdd(&b_assignedPoints[(int)d_dataPoints[tid]], 1);
	}

	__syncthreads();

	if (threadIdx.x == 0)
		for (int i = 0; i < k; i++) {
			atomicAdd(&assignedPoints[i], b_assignedPoints[i]);
			for (int j = 0; j < dim; j++)
				atomicAddDouble(&d_centroids[(i * dim) + j], b_centroids[(i * dim) + j]);
		}

	__syncthreads();

	if (threadIdx.x == 0) {
		atomicAdd(&countB, 1);
	}

	__syncthreads();

	if (tmp < k * dim) {
		while (countB != 2 * NumBlocks) {
		}
		if (assignedPoints[tmp / k] != 0)
			for (int i = tmp; i < tmp + dim; i++)
				d_centroids[i] /= assignedPoints[tmp / k];
		countB = 0;
	}
}

__global__ void k_means_cuda_device_assign_centroids(double* d_dataPoints, double* d_centroids, int length, int dim, int k, bool* d_convergenceCheck, int NumBlocks) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	tid *= (dim + 1);
	if (tid >= length * (dim + 1))
		return;
	int newCentroid = calculateCentroid_device(d_dataPoints, tid, dim, k, d_centroids);
	__syncthreads();
	if ((int)(d_dataPoints[tid + dim]) != newCentroid) {
		AtomicBool(&convergenceCheck, false);
		d_dataPoints[tid + dim] = newCentroid;
	}
	__syncthreads();

	if (threadIdx.x == 0)
		atomicAdd(&countB, 1);

	__syncthreads();
	if (tid == 0) {
		while (countB != NumBlocks) {
		}
		countB = 0;
		d_convergenceCheck[0] = convergenceCheck;
		convergenceCheck = true;
	}
}

void k_means_cuda_host(float** dataPoints, int length, int dim, bool useParallelism, int k, std::mt19937 seed) {
	// Randomizer
	std::uniform_real_distribution<> distrib(0, 10);

	double* h_dataPoints = new double[length * (dim + 1)];
	double* h_centroids = new double[k * dim];
	int* h_assignedPoints = new int[k];

	// 1. Choose the number of clusters(K) and obtain the data points
	if (k <= 0) {
		k = 1;
	}

	//2. Place the centroids c_1, c_2, .....c_k randomly
	float** centroids = new float* [k];
	for (int i = 0; i < k; i++) {
		centroids[i] = new float[dim];
		h_assignedPoints[i] = 0;

		// rand init
		for (int j = 0; j < dim; j++) {
			centroids[i][j] = distrib(seed);	// random clusters
		}
	}

	linealizer(h_dataPoints, dataPoints, length, dim + 1);
	linealizer(h_centroids, centroids, k, dim);

	double* d_dataPoints;

	double* d_centroids;

	bool* d_convergenceCheck;

	int* d_assignedPoints;

	int NumBlocks;

	bool convergence = false;

	bool* convergenceCheck = new bool[1];

	convergenceCheck[0] = true;

	HANDLE_ERROR(hipMalloc((void**)&d_dataPoints, sizeof(double) * length * (dim + 1)));
	HANDLE_ERROR(hipMalloc((void**)&d_centroids, sizeof(double) * k * dim));
	HANDLE_ERROR(hipMalloc((void**)&d_assignedPoints, sizeof(int) * k));
	HANDLE_ERROR(hipMalloc((void**)&d_convergenceCheck, sizeof(bool)));

	HANDLE_ERROR(hipMemcpy(d_dataPoints, h_dataPoints, sizeof(double) * length * (dim + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_centroids, h_centroids, sizeof(double) * k * dim, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_convergenceCheck, convergenceCheck, sizeof(bool), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_assignedPoints, h_assignedPoints, sizeof(int) * k, hipMemcpyHostToDevice));

	while (!convergence) {

		convergenceCheck[0] = true;

		NumBlocks = length / 256;

		if (length % 256 != 0) {
			NumBlocks += 1;
		}

		k_means_cuda_device_assign_centroids << < NumBlocks, 256 >> > (d_dataPoints, d_centroids, length, dim, k, d_convergenceCheck, NumBlocks);
		hipDeviceSynchronize();

		HANDLE_ERROR(hipMemcpy(convergenceCheck, d_convergenceCheck, sizeof(bool), hipMemcpyDeviceToHost));

		convergence = convergenceCheck[0];

		NumBlocks = (length * (dim + 1)) / 256;

		if ((length * (dim + 1)) % 256 != 0) {
			NumBlocks += 1;
		}

		k_means_cuda_device_update_centroids << < NumBlocks, 256, k* (dim + 1) * sizeof(double) + k * sizeof(int) >> > (d_dataPoints, d_centroids, d_assignedPoints, length, dim, k, NumBlocks);
		hipDeviceSynchronize();

	}

	HANDLE_ERROR(hipMemcpy(h_dataPoints, d_dataPoints, sizeof(double) * length * (dim + 1), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(h_centroids, d_centroids, sizeof(double) * k * dim, hipMemcpyDeviceToHost));

	/*printf("DATAPOINTS\n");
	int count = 1;
	for (int i = 0; i < length * (dim + 1); i++) {
		printf("%f ", h_dataPoints[i]);
		if (count == (dim+1)) {
			printf("\n");
			count = 0;
		}
		count++;
	}
	printf("CENTROIDS\n");
	for (int i = 0; i < k * dim; i++) {
		printf("%f ", h_centroids[i]);
		if (count == dim) {
			printf("\n");
			count = 0;
		}
		count++;
	}*/

	delinealizer(dataPoints, h_dataPoints, length * (dim + 1), dim);

	hipFree(d_dataPoints);
	hipFree(d_centroids);
	hipFree(d_assignedPoints);
	hipFree(d_convergenceCheck);
	free(h_dataPoints);
	free(h_centroids);
	free(h_assignedPoints);
	free(convergenceCheck);

}

/*
Handles CUDA Errors and print them.
*/
static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}