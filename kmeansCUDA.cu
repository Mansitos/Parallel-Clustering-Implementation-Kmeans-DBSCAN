#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <iostream>
#include "utils.h"

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char* file, int line);
__device__ float euclideanDistance(float* dataPoint, float* centroid, int dim);
__device__ int calculateCentroid(float* dataPoint, int dim, int k, float** centroids);

__device__ unsigned int countB = 0;			// counter for CUDA blocks
__device__ unsigned int lock = 0;			// variable to lock access to part of code
__device__ int convergenceThreshold = 0; 	// 0%
__device__ int errorsCounter = 0; 			// the amount of different results
__device__ int assigned = 0;				// counter of elements assigned 

/*
Atomic add implementation for double
	@address: the address of the value to be updatig
	@val: the value to be adding

	Return: the old value of address
*/
__device__ double atomicAddDouble(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

/*
Function for euclidean distance calculation between 2 given points
	@dataPoint: the list of points (1st points)
	@tid: the index of the first point
	@centroid: the list of centroids (2nd points)
	@index: the index of the second points
	@dim: dimension of points: 3D, 4D, etc.

	Return: the distance between dataPoint and centroid
*/
__device__ double euclideanDistance_device(double* dataPoint, int tid, double* centroid, int index, int dim) {
	double sum = 0;
	for (int i = 0; i < dim; i++) {
		sum += pow((dataPoint[tid + i] - centroid[index + i]), 2);
	}
	double distance = sqrt(sum);
	return distance;
}

/*
Function for assigning a centroid to a given point. Iterates each centroid and check which one is the closest one.
	@dataPoint: the list of points
	@tid: the index of the point for which centroid have to be calculated
	@dim: dimension of points: 3D etc.
	@k: number of centroids
	@centroids: list of centroids.

	Return: the index of the nearest centroid 
*/
__device__ int calculateCentroid_device(double* dataPoint, int tid, int dim, int k, double* centroids) {
	int nearestCentroidIndex = dataPoint[tid + dim];
	bool firstIteration = true;
	double minDistance = 0;
	for (int i = 0; i < k; i++) {
		double distance = euclideanDistance_device(dataPoint, tid, centroids, i * dim, dim);
		if (firstIteration) {
			firstIteration = false;
			nearestCentroidIndex = i;
			minDistance = distance;
		}
		else if (distance < minDistance) {
			nearestCentroidIndex = i;
			minDistance = distance;
		}
	}
	return nearestCentroidIndex;
}

/*
CUDA Kernel for parallel computing of updated centroids coordinates. (parallel division)
	@d_centroids: the list of centroids (device side)
	@assignedPoints: the amount of assigned points to that cluster
	@dim: dimension of points
	@index: the index of the cluster to update
*/
__global__ void computeCentroids(double* d_centroids, int assignedPoint, int dim, int index) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dim)
		return;
	d_centroids[index + tid] /= assignedPoint;
}

/*
CUDA Kernel for centroids update.
	@d_dataPoints: the list of points (device side)
	@d_centroids: the list of centroids (device side)
	@assignedPoints: the amount of assigned points to that cluster
	@dim: dimension of points
	@lenght: amount of points
	@k: number of centroids
	@NumBlocks: the number of blocks which are running this kernel
*/
__global__ void k_means_cuda_device_update_centroids(double* d_dataPoints, double* d_centroids, int* assignedPoints, int length, int dim, int k, int NumBlocks) {
	int threadsXblock = 1024;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= length * (dim + 1))
		return;

	if (threadIdx.x == 0)
		atomicAdd(&countB, 1);
	__syncthreads();

	extern __shared__ double s[];
	double* b_centroids = s;
	int* b_assignedPoints = (int*)&b_centroids[k * (dim + 1)];
	if (threadIdx.x < k)
		b_assignedPoints[threadIdx.x] = 0;
	int mul = 0;
	while (threadIdx.x + mul * threadsXblock < k * dim) {
		b_centroids[threadIdx.x + mul * threadsXblock] = 0;
		mul++;
	}
	__syncthreads();

	// Sum of point's coordinates for each cluster
	int clusterIndex = ((int)(tid / (dim + 1))) * (dim + 1) + dim;
	int coordOffest = tid % (dim + 1);

	if (tid != clusterIndex) {
		atomicAddDouble(&b_centroids[(int)d_dataPoints[clusterIndex] * dim + coordOffest], d_dataPoints[tid]);
	}
	else {
		atomicAdd(&b_assignedPoints[(int)d_dataPoints[tid]], 1);
	}

	__syncthreads();

	if (threadIdx.x < k)
		atomicAdd(&assignedPoints[threadIdx.x], b_assignedPoints[threadIdx.x]);
	mul = 0;
	while (threadIdx.x + mul * threadsXblock < k * dim) {
		atomicAddDouble(&d_centroids[threadIdx.x + mul * threadsXblock], b_centroids[threadIdx.x + mul * threadsXblock]);
		mul++;
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		atomicAdd(&countB, 1);
	}

	__syncthreads();

	if (countB >= 2 * NumBlocks) {
		if (threadIdx.x < k && atomicAdd(&lock, 1) < k) {
			int tmpIndex = atomicAdd(&assigned, 1);
			if (assignedPoints[tmpIndex] != 0) {
				int NumBlocksChild = dim / threadsXblock;
				if (dim % threadsXblock != 0)
					NumBlocksChild++;
				computeCentroids << <NumBlocksChild, threadsXblock >> > (d_centroids, assignedPoints[tmpIndex], dim, tmpIndex * dim);
				hipDeviceSynchronize();
			}
		}
	}

	__syncthreads();
	if (threadIdx.x == 0 && countB >= 2 * NumBlocks && assigned >= k) {
		assigned = 0;
		countB = 0;
		lock = 0;

	}

}

/*
For each point calculate the new centroid and assign it.
	@d_dataPoints: the list of points (device side)
	@d_centroids: the list of centroids (device side)
	@dim: dimension of points
	@length: amount of points
	@k: number of centroids
	@d_convergenceCheck: pointer of the convergence flag. If true the algorithm stops.
	@NumBlocks: the number of blocks which are running this kernel
*/
__global__ void k_means_cuda_device_assign_centroids(double* d_dataPoints, double* d_centroids, int length, int dim, int k, bool* d_convergenceCheck, int NumBlocks) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	tid *= (dim + 1);
	if (tid >= length * (dim + 1))
		return;
	int newCentroid = calculateCentroid_device(d_dataPoints, tid, dim, k, d_centroids);

	__syncthreads();

	if ((int)(d_dataPoints[tid + dim]) != newCentroid) {
		atomicAdd(&errorsCounter, 1);
		d_dataPoints[tid + dim] = newCentroid;
	}
	__syncthreads();

	if (threadIdx.x == 0) {
		atomicAdd(&countB, 1);
	}

	__syncthreads();

	if (countB >= NumBlocks) {
		if (threadIdx.x < 1024 && atomicAdd(&lock, 1) < 1) {
			countB = 0;
			lock = 0;
			if (errorsCounter <= ((length * convergenceThreshold) / 100)) {
				d_convergenceCheck[0] = true;
			}
			else {
				d_convergenceCheck[0] = false;
			}
			errorsCounter = 0;
		}
	}
}

/*
Main KMEANS call. Host initialization.
	@dataPoints: point to datapoints
	@length: number of points
	@dim: dimension of points (2D, 3D etc.)
	@useParallelism: to use or not OpenMP
	@seed: the seed for the input generator
*/
void k_means_cuda_host(float** dataPoints, int length, int dim, bool useParallelism, int k, std::mt19937 seed) {
	// Randomizer
	std::uniform_real_distribution<> distrib(0, length*100);

	double* h_dataPoints = new double[length * (dim + 1)];
	double* h_centroids = new double[k * dim];
	int* h_assignedPoints = new int[k];

	// 1. Choose the number of clusters(K) and obtain the data points
	if (k <= 0) {
		k = 1;
	}

	//2. Place the centroids c_1, c_2, .....c_k randomly
	float** centroids = new float* [k];
	for (int i = 0; i < k; i++) {
		centroids[i] = new float[dim];
		h_assignedPoints[i] = 0;

		// rand init
		for (int j = 0; j < dim; j++) {
			centroids[i][j] = distrib(seed);	// random clusters
		}
	}

	// Linearization of dataPoints (from nD to 1D) and centroids (from nD to 1D)
	linealizer(h_dataPoints, dataPoints, length, dim + 1);
	linealizer(h_centroids, centroids, k, dim);

	double* d_dataPoints;
	double* d_centroids;

	bool* d_convergenceCheck;
	int* d_assignedPoints;

	int NumBlocks;

	bool convergence = false;
	bool* convergenceCheck = (bool*)malloc(sizeof(bool));
	convergenceCheck[0] = true;

	// device allocation for linearized array
	HANDLE_ERROR(hipMalloc((void**)&d_dataPoints, sizeof(double) * length * (dim + 1)));
	HANDLE_ERROR(hipMalloc((void**)&d_centroids, sizeof(double) * k * dim));
	HANDLE_ERROR(hipMalloc((void**)&d_assignedPoints, sizeof(int) * k));
	HANDLE_ERROR(hipMalloc((void**)&d_convergenceCheck, sizeof(bool)));

	// copy of host linearized array to device
	HANDLE_ERROR(hipMemcpy(d_dataPoints, h_dataPoints, sizeof(double) * length * (dim + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_centroids, h_centroids, sizeof(double) * k * dim, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_convergenceCheck, convergenceCheck, sizeof(bool), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_assignedPoints, h_assignedPoints, sizeof(int) * k, hipMemcpyHostToDevice));

	int threadsXblock = 1024;	//number of thread per block
	//loop until convergence is not reached
	while (!convergence) {

		convergenceCheck[0] = true;
		NumBlocks = length / threadsXblock;

		if (length % threadsXblock != 0) {
			NumBlocks += 1;
		}
		//KMEANS call to assign centroids
		k_means_cuda_device_assign_centroids <<< NumBlocks, threadsXblock >>> (d_dataPoints, d_centroids, length, dim, k, d_convergenceCheck, NumBlocks);
		hipDeviceSynchronize();

		// copy of device convergence array to host
		HANDLE_ERROR(hipMemcpy(convergenceCheck, d_convergenceCheck, sizeof(bool), hipMemcpyDeviceToHost));

		convergence = convergenceCheck[0];
		NumBlocks = (length * (dim + 1)) / threadsXblock;

		if ((length * (dim + 1)) % threadsXblock != 0) {
			NumBlocks += 1;
		}

		//memset to zero for centroids and counter of assigned points in device copy
		HANDLE_ERROR(hipMemset(d_assignedPoints, 0, sizeof(int) * k));
		HANDLE_ERROR(hipMemset(d_centroids, 0, sizeof(double) * k * dim));

		//KMEANS call to update centroids
		k_means_cuda_device_update_centroids <<< NumBlocks, threadsXblock, k* (dim + 1) * sizeof(double) + k * sizeof(int) >>> (d_dataPoints, d_centroids, d_assignedPoints, length, dim, k, NumBlocks);
		hipDeviceSynchronize();

	}

	// copy of device linearized array (result) to host
	HANDLE_ERROR(hipMemcpy(h_dataPoints, d_dataPoints, sizeof(double) * length * (dim + 1), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(h_centroids, d_centroids, sizeof(double) * k * dim, hipMemcpyDeviceToHost));

	delinealizer(dataPoints, h_dataPoints, length * (dim + 1), dim);

	// Memory deallocation
	hipFree(d_dataPoints);
	hipFree(d_centroids);
	hipFree(d_assignedPoints);
	hipFree(d_convergenceCheck);
	free(h_dataPoints);
	free(h_centroids);
	free(h_assignedPoints);
	free(convergenceCheck);
}

/*
Handles CUDA Errors and print them.
*/
static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}